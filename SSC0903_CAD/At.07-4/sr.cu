/**
------------------------------------------------------
--- SSC0903 - Computação de Alto Desempenho (2020) ---
------------------------------------------------------

Atividade 07-4: Atividades em matrizes com CUDA

Desenvolvida por:
- Grupo 1
-- André Luís Mendes Fakhoury
-- Debora Buzon da Silva
-- Gustavo Vinicius Vieira Silva Soares
-- Thiago Preischadt Pinheiro
*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// numero de threads por bloco
#define BLOCK_SIZE 1024

// encontra o valor minimo e insere em v_min
__global__ void min_kernel(int *v, int N, int *v_min) {
    __shared__ int local_min;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (threadIdx.x == 0) {
        local_min = v[i];
    }
    __syncthreads();
    if (i < N) {
        atomicMin(&local_min, v[i]);
    }
    __syncthreads();
    if (threadIdx.x == 0) {
        atomicMin(v_min, local_min);
    }
}

// encontra o valor maximo e insere em v_max
__global__ void max_kernel(int *v, int n, int *v_max) {
    __shared__ int local_max;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (threadIdx.x == 0) {
        local_max = v[i];
    }
    __syncthreads();
    if (i < n) {
        atomicMax(&local_max, v[i]);
    }
    __syncthreads();
    if (threadIdx.x == 0) {
        atomicMax(v_max, local_max);
    }
}

// encontra a soma dos produtos escalares das linhas
__global__ void dot_kernel(int *a, int *b, int n, int *dot) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n * n) {
        int i = index / n;
        atomicAdd(&dot[i], a[index] * b[index]);
    }
}

//  soma os elementos de dot para dot[0]
__global__ void dot_sum_kernel(int *dot, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i > 0 && i < n) {
        atomicAdd(&dot[0], dot[i]);
    }
}

int main(int argc, char* argv[]) {
    if (argc < 2) {
        printf("Por favor, insira o arquivo a ser lido.");
        return -1;
    }

    FILE* input = fopen(argv[1], "r");
    if (input == NULL) {
        printf("Arquivo invalido.");
        return -1;
    }

    int n; // dimensao da matriz
    fscanf(input, "%d", &n);

    int N = n * n; // tamanho total

    int* a = (int*) malloc(N * sizeof(int));
    int* b = (int*) malloc(N * sizeof(int));

    // read matrix A
    for (int i = 0; i < N; i++) {
        fscanf(input, "%d", &a[i]);
    }

    // read matrix B
    for (int i = 0; i < N; i++) {
        fscanf(input, "%d", &b[i]);
    }

    // fecha o arquivo de entrada
    fclose(input);

    // -------------- fim da entrada -------------------------

    // inicializa os vetores CUDA
    int *a_cuda, *b_cuda;
    hipMalloc(&a_cuda, N * sizeof(int));
    hipMalloc(&b_cuda, N * sizeof(int));
    
    hipMemcpy(a_cuda, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(b_cuda, b, N * sizeof(int), hipMemcpyHostToDevice);
    
    // inicializa os vetores de resposta
    int *min, *max, *dot;
    hipHostMalloc(&min, sizeof(int), hipHostMallocDefault);
    hipHostMalloc(&max, sizeof(int), hipHostMallocDefault);
    hipHostMalloc(&dot, n * sizeof(int), hipHostMallocDefault); // um pra cada linha
    
    *min = *max = a[0];
    for (int i = 0; i < n; i++) {
        dot[i] = 0;
    }
    
    // aloca vetores das respostas na GPU
    int *min_cuda, *max_cuda, *dot_cuda;
    hipMalloc(&min_cuda, sizeof(int));
    hipMalloc(&max_cuda, sizeof(int));
    hipMalloc(&dot_cuda, n * sizeof(int));
    
    // cria as streams para cada operacao
    hipStream_t min_stream, max_stream, dot_stream;
    hipStreamCreate(&min_stream);
    hipStreamCreate(&max_stream);
    hipStreamCreate(&dot_stream);
    
    // colocando as operaceos de copia nas streams
    hipMemcpyAsync(min_cuda, min, sizeof(int), hipMemcpyHostToDevice, min_stream);
    hipMemcpyAsync(max_cuda, max, sizeof(int), hipMemcpyHostToDevice, max_stream);
    hipMemcpyAsync(dot_cuda, dot, n * sizeof(int), hipMemcpyHostToDevice, dot_stream);
    
    // chama os kernels de min
    min_kernel<<<(N + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE, 0, min_stream>>>(a_cuda, N, min_cuda);
    min_kernel<<<(N + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE, 0, min_stream>>>(b_cuda, N, min_cuda);
    
    // chama os kernels de max
    max_kernel<<<(N + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE, 0, max_stream>>>(a_cuda, N, max_cuda);
    max_kernel<<<(N + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE, 0, max_stream>>>(b_cuda, N, max_cuda);
    
    // chama os kernels de produto escalar
    dot_kernel<<<(N + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE, 0, dot_stream>>>(a_cuda, b_cuda, n, dot_cuda);

    // chama os kernels de somar os produtos escalares
    dot_sum_kernel<<<(n + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE, 0, dot_stream>>>(dot_cuda, n);
    
    // copia as respostas de volta
    hipMemcpyAsync(min, min_cuda, sizeof(int), hipMemcpyDeviceToHost, min_stream);
    hipMemcpyAsync(max, max_cuda, sizeof(int), hipMemcpyDeviceToHost, max_stream);
    hipMemcpyAsync(dot, dot_cuda, n * sizeof(int), hipMemcpyDeviceToHost, dot_stream);
    
    // sincroniza as streams
    hipStreamSynchronize(min_stream);
    hipStreamSynchronize(max_stream);
    hipStreamSynchronize(dot_stream);
    
    // destroi as streams
    hipStreamDestroy(min_stream);
    hipStreamDestroy(max_stream);
    hipStreamDestroy(dot_stream);
    
    // imprime a resposta
    printf("%d %d %d\n", dot[0], *min, *max);
    
    // libera as memorias utilizadas
    free(a);
    free(b);
    
    hipFree(a_cuda);
    hipFree(b_cuda);
    hipFree(min_cuda);
    hipFree(max_cuda);
    hipFree(dot_cuda);
    
    hipHostFree(min);
    hipHostFree(max);
    hipHostFree(dot);

    return 0;
}
